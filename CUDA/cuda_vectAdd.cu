/* File:vectAdd.c
 Compile:
    gcc -o add vectAdd.c -lm
 Run:
    ./add <vector dimension>
 
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include<sys/time.h>
#include<math.h>

/* Function that adds two vectors */
__global__ void addVectors(double *x, double *y, double*z, int n){
    int j = threadIdx.x + blockDim.x * blockIdx.x;
    if(j<n){
        z[j] = x[j] + y[j];
    }
}

/*--------------------------------
 Function to Print Vectors
 --------------------------------*/
void printVectors(double *a, double *b, double*c, int n) {
    int k = 0;
    for(k=0;k<n;k++)
    printf("\n %f + %f  = %f",  a[k] , b[k], c[k]);
}

/*--------------------------------
 Function to measure run time
 --------------------------------*/
double getTime(){
    struct timeval t;
    gettimeofday(&t, NULL);
    return t.tv_sec + t.tv_usec/1000000.0;
    
}

/*------------------------------------------------------------------*/
int main(int argc, char* argv[]) {
    
    double start, elapsed;
    double* z; double* x; double* y;
    int n;
    int i;
    n=atoi(argv[1]); // get vector dimension
    hipMallocManaged(&x,n*sizeof(double));
   hipMallocManaged(&y,n*sizeof(double));
    hipMallocManaged(&z,n*sizeof(double));
    
    // generate vectors
    for (i=0;i<n;i++){
        x[i]=sin(i)*sin(i);
        y[i]=cos(i)*cos(i);
    }
    start = getTime();
     
    int blockSize = 256;
     int numBlocks = ceil((double)n/blockSize);
    addVectors<<<numBlocks,blockSize>>>(x, y, z,  n);
    hipDeviceSynchronize();
    elapsed = getTime() - start;
    printf("\nN=%d; numBlocks: %d ,blockSize : %d, Elapsed time:%f\n",n,numBlocks,blockSize,elapsed);
    // Print vectors
    if (n<21){
        printVectors(x, y, z,  n);
    }
    // Check if vector z is correct, this should equal 1
    double sum = 0;
    for(i=0; i<n; i++){
        sum += z[i];
    }
    printf("final result: %f\n", sum/n);
    hipFree(x); hipFree(y); hipFree(z);
    return 0;
}  /* main */



